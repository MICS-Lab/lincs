#include "hip/hip_runtime.h"
// Copyright 2021 Vincent Jacques

#include "randomness.hpp"

__global__ void initialize_rng(hiprandState* const rng_states, const uint seed) {
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, tid, 0, &rng_states[tid]);
}
