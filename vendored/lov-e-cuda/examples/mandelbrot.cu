// Copyright 2014 Andy Adinets
// Copyright 2022 Vincent Jacques
// Copyright 2022 Laurent Cabaret
// File copied from https://github.com/canonizer/mandelbrot-dyn
// (https://developer.nvidia.com/blog/introduction-cuda-dynamic-parallelism/)
// then modified by Vincent Jacques to fit this project's coding guidelines


#include <hip/hip_runtime.h>
#include <png.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>


#define H (16 * 1024)
#define W (16 * 1024)
#define IMAGE_PATH "./mandelbrot.png"
#define MAX_DWELL 512
#define CUT_DWELL (MAX_DWELL / 4)
// Block size along
#define BSX 64
#define BSY 4


// Get the color, given the dwell (on host)
void dwell_color(int* r, int* g, int* b, int dwell) {
  // Black for the Mandelbrot set
  if (dwell >= MAX_DWELL) {
    *r = *g = *b = 0;
  } else {
    // Cut at zero
    if (dwell < 0)
      dwell = 0;
    if (dwell <= CUT_DWELL) {
      // From black to blue the first half
      *r = *g = 0;
      *b = 128 + dwell * 127 / (CUT_DWELL);
    } else {
      // From blue to white for the second half
      *b = 255;
      *r = *g = (dwell - CUT_DWELL) * 255 / (MAX_DWELL - CUT_DWELL);
    }
  }
}


void save_image(const char* filename, int* dwells, unsigned w, unsigned h) {
  // Code taken from http://www.labbookpages.co.uk/software/imgProc/libPNG.html
  png_bytep row;
  FILE* fp = fopen(filename, "wb");
  png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, 0, 0, 0);
  png_infop info_ptr = png_create_info_struct(png_ptr);
  png_init_io(png_ptr, fp);
  png_set_IHDR(
    png_ptr, info_ptr, w, h, 8, PNG_COLOR_TYPE_RGB,
    PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);
  png_text title_text;
  title_text.compression = PNG_TEXT_COMPRESSION_NONE;
  title_text.key = const_cast<char*>("Title");
  title_text.text = const_cast<char*>("Mandelbrot set, per-pixel");
  png_set_text(png_ptr, info_ptr, &title_text, 1);
  png_write_info(png_ptr, info_ptr);
  row = reinterpret_cast<png_bytep>(malloc(3 * w * sizeof(png_byte)));
  for (unsigned y = 0; y < h; y++) {
    for (unsigned x = 0; x < w; x++) {
      int r, g, b;
      dwell_color(&r, &g, &b, dwells[y * w + x]);
      row[3 * x + 0] = (png_byte)r;
      row[3 * x + 1] = (png_byte)g;
      row[3 * x + 2] = (png_byte)b;
    }
    png_write_row(png_ptr, row);
  }
  png_write_end(png_ptr, nullptr);

  fclose(fp);
  png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
  png_destroy_write_struct(&png_ptr, nullptr);
  free(row);
}


struct complex {
  __host__ __device__ complex(const float re_, const float im_ = 0) : re(re_), im(im_) {}

  float re, im;
};

inline __host__ __device__ complex operator+(const complex &a, const complex &b) {
  return complex(a.re + b.re, a.im + b.im);
}

inline __host__ __device__ complex operator-(const complex &a) {
  return complex(-a.re, -a.im);
}

inline __host__ __device__ complex operator-(const complex &a, const complex &b) {
  return complex(a.re - b.re, a.im - b.im);
}

inline __host__ __device__ complex operator*(const complex &a, const complex &b) {
  return complex(a.re * b.re - a.im * b.im, a.im * b.re + a.re * b.im);
}

inline __host__ __device__ float abs2(const complex &a) {
  return a.re * a.re + a.im * a.im;
}

inline __host__ __device__ complex operator/(const complex &a, const complex &b) {
  const float invabs2 = 1 / abs2(b);
  return complex((a.re * b.re + a.im * b.im) * invabs2,
                 (a.im * b.re - b.im * a.re) * invabs2);
}


// Check CUDA errors
#define cucheck(call) { \
  hipError_t res = (call); \
  if (res != hipSuccess) { \
    const char* err_str = hipGetErrorString(res); \
    fprintf(stderr, "%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call); \
    exit(-1); \
  } \
}


int divup(int x, int y) { return x / y + (x % y ? 1 : 0); }

// Compute the dwell for a single pixel
__device__
int pixel_dwell(int w, int h, complex cmin, complex cmax, int x, int y) {
  const complex dc = cmax - cmin;
  const float fx = static_cast<float>(x) / w;
  const float fy = static_cast<float>(y) / h;
  const complex c = cmin + complex(fx * dc.re, fy * dc.im);
  int dwell = 0;
  complex z = c;
  while (dwell < MAX_DWELL && abs2(z) < 2 * 2) {
    z = z * z + c;
    dwell++;
  }
  return dwell;
}

/*
Compute the dwells for Mandelbrot image

@param dwells the output array
@param w the width of the output image
@param h the height of the output image
@param cmin the complex value associated with the bottom-left corner of the image
@param cmax the complex value associated with the top-right corner of the image
*/
__global__
void mandelbrot_k(int* dwells, int w, int h, complex cmin, complex cmax) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  dwells[y * w + x] = pixel_dwell(w, h, cmin, cmax, x, y);
}

int main(int, char*[]) {
  const int w = W;
  const int h = H;

  const size_t dwell_sz = w * h * sizeof(int);
  int* d_dwells;
  cucheck(hipMalloc(reinterpret_cast<void**>(&d_dwells), dwell_sz));
  int* const h_dwells = reinterpret_cast<int*>(malloc(dwell_sz));

  const dim3 threads(BSX, BSY);
  const dim3 blocks(divup(w, threads.x), divup(h, threads.y));

  const double t1 = omp_get_wtime();
  mandelbrot_k<<<blocks, threads>>>(d_dwells, w, h, complex(-1.5, -1), complex(0.5, 1));
  cucheck(hipDeviceSynchronize());
  const double t2 = omp_get_wtime();

  cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
  save_image(IMAGE_PATH, h_dwells, w, h);

  const double gpu_time = t2 - t1;
  printf("Mandelbrot set computed in %.3lf s, at %.3lf Mpix/s\n", gpu_time, h * w * 1e-6 / gpu_time);

  free(h_dwells);
  hipFree(d_dwells);
}
